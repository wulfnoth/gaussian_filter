#include "hip/hip_runtime.h"


#include <stdio.h>
#include <math.h>
#include <iostream>

using namespace std;

#define BLOCKDIM_X		16
#define BLOCKDIM_Y		16

#define GRIDDIM_X		256
#define GRIDDIM_Y		256
#define MASK_WIDTH		5

__constant__ int d_const_Gaussian[MASK_WIDTH*MASK_WIDTH]; //常量

static __global__ void kernel_GaussianFilt(int width, int height, int byte_per_pixel, unsigned char *d_src_imgbuf, unsigned char *d_guassian_imgbuf);
int parseInt(int , char* );
int read(FILE*, int, int);
int** parse_bmp(const char* filepath, int* width, int* height);
void write_buffer(int value, FILE* file, int length);
void write_file(const char* filepath, int width, int height, int** data);
unsigned char * transformToUCharVector(int ** data, int width, int height, int byte_per_pixel);
int ** transformToIntMatrix(unsigned char * data, int width, int height, int byte_per_pixel);

unsigned long GetTickCount()
{
    struct timespec ts;

    clock_gettime(CLOCK_MONOTONIC, &ts);

    return (ts.tv_sec * 1000 + ts.tv_nsec / 1000000);

}


int main(int argc, char **argv)
{
	char* input_path;
	char* output_path;
	if (argc != 3) {
		input_path = "/home/cloud/workspace/hand/test1.bmp";
		output_path = "/home/cloud/workspace/hand/result.bmp";
	} else {
		input_path = argv[1];
		output_path = argv[2];
	}

	// char * input_path = "C:\\Users\\congj\\Desktop\\result.bmp";
	// char * output_path = "C:\\Users\\congj\\Desktop\\gs.bmp";
	printf("input_path: %s\n", input_path);
	printf("output_path: %s\n", output_path);
	
	struct hipDeviceProp_t pror;
	hipGetDeviceProperties(&pror, 0);
	cout << "maxThreadsPerBlock=" << pror.maxThreadsPerBlock << endl;

	long start, end;
	long time = 0;

	start = GetTickCount();
	hipEvent_t startt, stop; 
	hipEventCreate(&startt);
	hipEventCreate(&stop);
	hipEventRecord(startt, 0);

	unsigned char *h_src_imgbuf;
	int width, height, byte_per_pixel = 3;

	int **d = parse_bmp(input_path, &width, &height);
	h_src_imgbuf = transformToUCharVector(d, width, height, byte_per_pixel);

	printf("width: %d, height: %d, byte_per_pixel: %d\n", width, height, byte_per_pixel);

	int size1 = width * height *byte_per_pixel * sizeof(unsigned char);

	//host memory
	unsigned char *h_guassian_imgbuf = new unsigned char[width*height*byte_per_pixel];

	//device memory
	unsigned char *d_src_imgbuf;
	unsigned char *d_guassian_imgbuf;

	hipMalloc((void**)&d_src_imgbuf, size1);
	hipMalloc((void**)&d_guassian_imgbuf, size1);

	//copy data from host to device
	hipMemcpy(d_src_imgbuf, h_src_imgbuf, size1, hipMemcpyHostToDevice);

	//gaussian matrix constant memory
	int Gaussian[25] = { 1,4,7,4,1,
						4,16,26,16,4,
						7,26,41,26,7,
						4,16,26,16,4,
						1,4,7,4,1 };//sum is 273
	hipMemcpyToSymbol(HIP_SYMBOL(d_const_Gaussian), Gaussian, 25 * sizeof(int));

	int bx = ceil((double)width / BLOCKDIM_X); // 40
	int by = ceil((double)height / BLOCKDIM_Y); //26

	if (bx > GRIDDIM_X) bx = GRIDDIM_X;
	if (by > GRIDDIM_Y) by = GRIDDIM_Y;
	//suppose width=638, height=411

	dim3 grid(bx, by); //40,26
	dim3 block(BLOCKDIM_X, BLOCKDIM_Y); //16,16

	//kernel
	kernel_GaussianFilt<<<grid, block>>>(width, height, byte_per_pixel, d_src_imgbuf, d_guassian_imgbuf);
	hipMemcpy(h_guassian_imgbuf, d_guassian_imgbuf, size1, hipMemcpyDeviceToHost);

	// saveBmp(output_path, h_guassian_imgbuf, width, height, byte_per_pixel);
	write_file(output_path, width, height, transformToIntMatrix(h_guassian_imgbuf, width, height, byte_per_pixel));
	//
	hipFree(d_src_imgbuf);
	hipFree(d_guassian_imgbuf);

	delete[]h_src_imgbuf;
	delete[]h_guassian_imgbuf;

	end = GetTickCount();
	//InterlockedExchangeAdd(&time, end - start); //window api
	__sync_fetch_and_add(&time, end - start); // linux api
	cout << "Total time GPU:";
	cout << time << endl;

	return 0;
}

static __global__ void kernel_GaussianFilt(int width, int height, int byte_per_pixel, unsigned char *d_src_imgbuf, unsigned char *d_dst_imgbuf)
{
	const int tix = blockDim.x * blockIdx.x + threadIdx.x;
	const int tiy = blockDim.y * blockIdx.y + threadIdx.y;
	/*cout << threadIdx.x << endl;
	cout << threadIdx.y << endl;*/
	const int threadTotalX = blockDim.x * gridDim.x;
	const int threadTotalY = blockDim.y * gridDim.y;

	for (int ix = tix; ix < height; ix += threadTotalX)
		for (int iy = tiy; iy < width; iy += threadTotalY)
		{
			for (int k = 0; k < byte_per_pixel; k++)
			{
				int sum = 0;
				int tempPixelValue = 0;
				for (int m = -2; m <= 2; m++)
				{
					for (int n = -2; n <= 2; n++)
					{
						if (ix + m < 0 || iy + n < 0 || ix + m >= height || iy + n >= width)
							tempPixelValue = 0;
						else
							tempPixelValue = *(d_src_imgbuf + (ix + m)*width*byte_per_pixel + (iy + n)*byte_per_pixel + k);
						sum += tempPixelValue * d_const_Gaussian[(m + 2) * 5 + n + 2];
					}
				}

				if (sum / 273 < 0)
					*(d_dst_imgbuf + (ix)*width*byte_per_pixel + (iy)*byte_per_pixel + k) = 0;
				else if (sum / 273 > 255)
					*(d_dst_imgbuf + (ix)*width*byte_per_pixel + (iy)*byte_per_pixel + k) = 255;
				else
					*(d_dst_imgbuf + (ix)*width*byte_per_pixel + (iy)*byte_per_pixel + k) = sum / 273;
			}
		}
}

int parseInt(int length, char* s)
{
	int result = 0;
	int shift = 0;
	for (int i = 0; i < length; i++)
	{
		//cout << hex << (int)(s[i] & 0x000000ff) << endl;
		result += (s[i]& 0x000000ff) << shift;
		shift += 8;
	}
	return result;
}

int read(FILE* file, int offset, int length)
{
	static char buff[4];
	fseek(file, offset, 0);
	fread(buff, sizeof(char), length, file);
	//current = offset + length;

	//cout << "current: " << ftell(file) << endl;
	return parseInt(length, buff);
}

int** parse_bmp(const char* filepath, int* width, int* height)
{
	FILE * file = fopen(filepath, "rb");
	if (!file) 
	{
		cerr << "文件打开失败。" << endl;
		exit(-1);
	}
	fseek(file, 0x0A, 0);

	int content_offset = read(file, 0x0A, 4);
	*width = read(file, 0x12, 4);
	*height = read(file, 0x16, 4);

	int** result;
	result = (int**)malloc(sizeof(int*) * 3);
	for (int i = 0; i < 3; i++)
	{
		result[i] = (int*)malloc(sizeof(int) * (*width) * (*height));
	}

	fseek(file, content_offset, 0);

	int pixel_acount = (*width) * (*height);

	int byte_in_row = *width * 24/8;
	int actual_byte_in_row = byte_in_row + 4 - byte_in_row % 4;

	cout << byte_in_row << endl;
	cout << actual_byte_in_row << endl;

	char* buffer;
	buffer = (char *)malloc(sizeof(char) * actual_byte_in_row);
	for (int i = 0; i < *height; i++) {
		fread(buffer, sizeof(char), actual_byte_in_row, file);
		for (int j = 0; j < *width; j++) {
			result[0][i* *width + j] = buffer[3 * j] & 0x000000ff;
			result[1][i* *width + j] = buffer[3 * j + 1] & 0x000000ff;
			result[2][i* *width + j] = buffer[3 * j + 2] & 0x000000ff;
		}
	}
	

	fclose(file);
	return result;
}

void write_buffer(int value, FILE* file, int length)
{
	static char buffer[4];
	for (int i = 0; i < length; i++)
	{
		char v_low8 = value & 0x000000ff;
		//cout << hex << int(v_low8) << endl;
		value = value >> 8;
		buffer[i] = v_low8;
	}
	for (int i = length - 1; i >= 0; i--)
		fwrite(buffer+length-1-i, sizeof(char), 1, file);
}

void write_file(const char* filepath, int width, int height, int** data)
{
	FILE *file = fopen(filepath, "wb");
	if (!file)
	{
		cerr << "文件打开错误" << endl;
		exit(-1);
	}
	char buffer[4];
	buffer[0] = 0x42;
	buffer[1] = 0x4D;
	fwrite(buffer, sizeof(char), 2, file); //写入BM

	int byte_in_row = width * 24 / 8;
	int actual_byte_in_row = byte_in_row + 4 - byte_in_row % 4;

	int size = 54 + actual_byte_in_row*height;
	write_buffer(size, file, 4); //写入文件大小的字节数
	write_buffer(0, file, 2); //写入保留字节 2个字节
	write_buffer(0, file, 2); //写入保留字节 2个字节
	write_buffer(54, file, 4); //写入偏移量，4个字节
	write_buffer(40, file, 4); //写入头部长度
	write_buffer(width, file, 4); //写入宽度
	write_buffer(height, file, 4); //写入高度
	write_buffer(1, file, 2); //平面数，总是被设置为1
	write_buffer(24, file, 2); //每像素位数
	write_buffer(0, file, 4); //不压缩
	write_buffer(height*actual_byte_in_row, file, 4); //图像字节数
	write_buffer(0, file, 4); //图像字节数
	write_buffer(0, file, 4); //图像字节数
	write_buffer(0, file, 4); //图像字节数
	write_buffer(0, file, 4); //图像字节数

	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++)
		{
			write_buffer(data[0][i*width + j], file, 1);
			write_buffer(data[1][i*width + j], file, 1);
			write_buffer(data[2][i*width + j], file, 1);
		}
		if (actual_byte_in_row - byte_in_row != 0)
			write_buffer(0, file, actual_byte_in_row - byte_in_row);
		cout << i << endl;
	}
	

	fclose(file);
}

unsigned char * transformToUCharVector(int ** data, int width, int height, int byte_per_pixel)
{
	unsigned char * result;
	result = (unsigned char *)malloc(sizeof(unsigned char) * width * height * byte_per_pixel);
	for (int i=0; i<width*height; i++)
	{
		for (int j=0; j<byte_per_pixel; j++)
		{
			result[i*byte_per_pixel + j] = (unsigned char)(data[j][i] & 0x000000ff);
		}
	}
	return result;
}

int ** transformToIntMatrix(unsigned char * data, int width, int height, int byte_per_pixel)
{
	int ** result;
	result = (int**)malloc(sizeof(int *)*byte_per_pixel);
	
	for (int i=0; i<byte_per_pixel; i++)
		result[i] = (int *)malloc(sizeof(int)*width*height);
	
	for (int i=0; i<width*height; i++)
		for (int j=0; j<byte_per_pixel; j++) 
			result[j][i] = data[i*byte_per_pixel+j] & 0x000000ff;
	
	return result;
}
